#include "hip/hip_runtime.h"
#include <libavutil/frame.h>
#include "hip/hip_runtime.h"
#include "VideoProcessor.h"

__device__ int calculateBillinearInterpolation(unsigned char* data, float x, float y, int xDiff, int yDiff, int linesize, int width, int height, float weightX, float weightY) {
	int startIndex = x + y * linesize;
	if (x + xDiff >= width)
		xDiff = 0;
	if (y + yDiff >= height)
		linesize = 0;
	int A = data[startIndex];
	int B = data[startIndex + xDiff];
	int C = data[startIndex + linesize * yDiff];
	int D = data[startIndex + linesize * yDiff + xDiff];

	//the most precise one
	int value = (int)(
		A * (1 - weightX) * (1 - weightY) +
		B * (weightX) * (1 - weightY) +
		C * (weightY) * (1 - weightX) +
		D * (weightX  *      weightY)
		);
	
	return value;
}

__device__ int calculateBicubicSplineInterpolation(unsigned char* data, int x, int y, int xDiff, int yDiff, int linesize, int width, int height, double weightX, double weightY) {
	int startIndex = x + y * linesize;
	int xDiffTop = xDiff;
	int yDiffTop = yDiff;

	if (x + xDiff >= width)
		xDiff = 0;
	if (x + xDiff * 2 >= width)
		xDiff = 0;
	if (x - xDiffTop < 0)
		xDiffTop = 0;
	if (y + yDiff >= height)
		yDiff = 0;
	if (y + yDiff * 2 >= height)
		yDiff = 0;
	if (y - yDiffTop < 0)
		yDiffTop = 0;

	double a = -0.75;
	double a0, a1, a2, a3;
	a0 = (a * weightX - 2 * a * pow(weightX, 2) + a * pow(weightX, 3)) * data[startIndex - xDiffTop - linesize * yDiffTop];
	a1 = (1 - (a + 3) * pow(weightX, 2) + (a + 2) * pow(weightX, 3)) * data[startIndex - linesize * yDiffTop];
	a2 = (-a * weightX + (2 * a + 3) * pow(weightX, 2) - (a + 2) * pow(weightX, 3)) * data[startIndex + xDiff - linesize * yDiffTop];
	a3 = (a * pow(weightX, 2) - a * pow(weightX, 3)) * data[startIndex + 2 * xDiff - linesize * yDiffTop];
	int b0 = round(a0 + a1 + a2 + a3);
	b0 = min(b0, 255);
	b0 = max(b0, 0);

	a0 = (a * weightX - 2 * a * pow(weightX, 2) + a * pow(weightX, 3)) * data[startIndex - xDiffTop];
	a1 = (1 - (a + 3) * pow(weightX, 2) + (a + 2) * pow(weightX, 3)) * data[startIndex];
	a2 = (-a * weightX + (2 * a + 3) * pow(weightX, 2) - (a + 2) * pow(weightX, 3)) * data[startIndex + xDiff];
	a3 = (a * pow(weightX, 2) - a * pow(weightX, 3)) * data[startIndex + 2 * xDiff];
	int b1 = round(a0 + a1 + a2 + a3);

	b1 = min(b1, 255);
	b1 = max(b1, 0);

	a0 = (a * weightX - 2 * a * pow(weightX, 2) + a * pow(weightX, 3)) * data[startIndex - xDiffTop + linesize * yDiff];
	a1 = (1 - (a + 3) * pow(weightX, 2) + (a + 2) * pow(weightX, 3)) * data[startIndex + linesize * yDiff];
	a2 = (-a * weightX + (2 * a + 3) * pow(weightX, 2) - (a + 2) * pow(weightX, 3)) * data[startIndex + xDiff + linesize * yDiff];
	a3 = (a * pow(weightX, 2) - a * pow(weightX, 3)) * data[startIndex + 2 * xDiff + linesize * yDiff];
	int b2 = round(a0 + a1 + a2 + a3);

	b2 = min(b2, 255);
	b2 = max(b2, 0);

	a0 = (a * weightX - 2 * a * pow(weightX, 2) + a * pow(weightX, 3)) * data[startIndex - xDiffTop + 2 * linesize * yDiff];
	a1 = (1 - (a + 3) * pow(weightX, 2) + (a + 2) * pow(weightX, 3)) * data[startIndex + 2 * linesize * yDiff];
	a2 = (-a * weightX + (2 * a + 3) * pow(weightX, 2) - (a + 2) * pow(weightX, 3)) * data[startIndex + xDiff + 2 * linesize * yDiff];
	a3 = (a * pow(weightX, 2) - a * pow(weightX, 3)) * data[startIndex + 2 * xDiff + 2 * linesize * yDiff];
	int b3 = round(a0 + a1 + a2 + a3);

	b3 = min(b3, 255);
	b3 = max(b3, 0);

	a0 = (a * weightY - 2 * a * pow(weightY, 2) + a * pow(weightY, 3)) * b0;
	a1 = (1 - (a + 3) * pow(weightY, 2) + (a + 2) * pow(weightY, 3)) * b1;
	a2 = (-a * weightY + (2 * a + 3) * pow(weightY, 2) - (a + 2) * pow(weightY, 3)) * b2;
	a3 = (a * pow(weightY, 2) - a * pow(weightY, 3)) * b3;
	int value = round(a0 + a1 + a2 + a3);
	value = min(value, 255);
	value = max(value, 0);

	return value;
}

__device__ int calculateBicubicPolynomInterpolation(unsigned char* data, float x, float y, int xDiff, int yDiff, int linesize, int width, int height, float weightX, float weightY) {
	int startIndex = x + y * linesize;
	x = weightX;
	y = weightY;

	if (x + xDiff >= width)
		xDiff = 0;
	if (x - xDiff < 0)
		xDiff = 0;
	if (y + yDiff >= height)
		yDiff = 0;
	if (y - yDiff < 0)
		yDiff = 0;

	double value;
	double p1  = data[startIndex                           ]; //f(y, x) = f1(0, 0)
	double b1 = (double)(x - 1) * (x - 2) * (x + 1) * (y - 1) * (y - 2) * (y + 1) / 4;
	value += p1 * b1;
	p1  = data[startIndex + xDiff                           ]; //f(y, x) = f2(0, 1)
	b1 = (double)-x * (x + 1) * (x - 2) * (y - 1) * (y - 2) * (y + 1) / 4;
	value += p1 * b1;
	p1  = data[startIndex              + linesize * yDiff   ]; //f(y, x) = f3(1, 0)
	b1 = (double)-y * (x - 1) * (x - 2) * (x + 1) * (y + 1) * (y - 2) / 4;
	value += p1 * b1;
	p1  = data[startIndex + xDiff      + linesize * yDiff   ]; //f(y, x) = f4(1, 1)
	b1 = (double)x * y * (x + 1) * (x - 2) * (y + 1) * (y - 2) / 4;
	value += p1 * b1;
	p1  = data[startIndex - xDiff                           ]; //f(y, x) = f5(0, -1)
	b1 = (double)-x * (x - 1) * (x - 2) * (y - 1) * (y - 2) * (y + 1) / 12;
	value += p1 * b1;
	p1  = data[startIndex             - linesize * yDiff    ]; //f(y, x) = f6(-1, 0)
	b1 = (double)-y * (x - 1) * (x - 2) * (x + 1) * (y - 1) * (y - 2) / 12;
	value += p1 * b1;
	p1  = data[startIndex - xDiff     + linesize * yDiff    ]; //f(y, x) = f7(1, -1)
	b1 = (double)x * y * (x - 1) * (x - 2) * (y + 1) * (y - 2) / 12;
	value += p1 * b1;
	p1  = data[startIndex + xDiff     - linesize * yDiff    ]; //f(y, x) = f8(-1, 1)
	b1 = (double)x * y * (x + 1) * (x - 2) * (y - 1) * (y - 2) / 12;
	value += p1 * b1;
	p1  = data[startIndex + 2 * xDiff                       ]; //f(y, x) = f9(0, 2)
	b1 = (double)x * (x - 1) * (x + 1) * (y - 1) * (y - 2) * (y + 1) / 12;
	value += p1 * b1;
	p1 = data[startIndex              + 2 * linesize * yDiff]; //f(y, x) = f10(2, 0)
	b1 = (double)y * (x - 1) * (x - 2) * (x + 1) * (y - 1) * (y + 1) / 12;
	value += p1 * b1;
	p1 = data[startIndex - xDiff      - linesize * yDiff    ]; //f(y, x) = f11(-1, -1)
	b1 = (double)x * y * (x - 1) * (x - 2) * (y - 1) * (y - 2) / 36;
	value += p1 * b1;
	p1 = data[startIndex + 2 * xDiff  + linesize * yDiff    ]; //f(y, x) = f12(1, 2)
	b1 = (double)-x * y * (x - 1) * (x + 1) * (y + 1) * (y - 2) / 12;
	value += p1 * b1;
	p1 = data[startIndex + xDiff      + 2 * linesize * yDiff]; //f(y, x) = f13(2, 1)
	b1 = (double)-x * y * (x + 1) * (x - 2) * (y - 1) * (y + 1) / 12;
	value += p1 * b1;
	p1 = data[startIndex + 2 * xDiff  - linesize * yDiff    ]; //f(y, x) = f14(-1, 2)
	b1 = (double)-x * y * (x - 1) * (x + 1) * (y - 1) * (y - 2) / 36;
	value += p1 * b1;
	p1 = data[startIndex - xDiff     + 2 * linesize * yDiff]; //f(y, x) = f15(2, -1)
	b1 = (double)-x * y * (x - 1) * (x - 2) * (y - 1) * (y + 1) / 36;
	value += p1 * b1;
	p1 = data[startIndex + 2 * xDiff + 2 * linesize * yDiff]; //f(y, x) = f16(2, 2)
	b1 = (double)  x * y * (x - 1) * (x + 1) * (y - 1) * (y + 1) / 36;
	value += p1 * b1;

	return value;
}

__device__ int calculateAreaInterpolation(unsigned char* data, int startIndex, float scaleX, float scaleY, int linesize, int stride, float* patternX, float* patternY) {
	float colorSum = 0;
	int rScaleX = ceilf(scaleX);
	int rScaleY = ceilf(scaleY);
	float divide = 0;
	for (int i = 0; i < rScaleY; i++) {
		for (int j = 0; j < rScaleX; j++) {
			int index = startIndex + j * stride + i * linesize;
			float weightX = patternX[j];
			float weightY = patternY[i];
			float weight = weightX * weightY;
			divide += weight;
			colorSum += (float)data[index] * weight;
		}
	}

	colorSum /= divide;
	return colorSum;
}

__global__ void resizeNV12DownscaleAreaKernel(unsigned char* inputY, unsigned char* inputUV, unsigned char* outputY, unsigned char* outputUV,
	int srcWidth, int srcHeight, int srcLinesizeY, int srcLinesizeUV, int dstWidth, int dstHeight, float xRatio, float yRatio, 
	float** patternX, int patternXSize, float** patternY, int patternYSize) {
	unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; //coordinate of pixel (y) in destination image
	unsigned int j = blockIdx.x * blockDim.x + threadIdx.x; //coordinate of pixel (x) in destination image
	if (i < dstHeight && j < dstWidth) {
		float yF = (int)(yRatio * i); //it's coordinate of pixel in source image
		float xF = (int)(xRatio * j); //it's coordinate of pixel in source image
		//bit to bit with above approach
		//float yF = (float)((i + 0.5f) * yRatio - 0.5f); //it's coordinate of pixel in source image
		//float xF = (float)((j + 0.5f) * xRatio - 0.5f); //it's coordinate of pixel in source image

		int x = floorf(xF);
		int y = floorf(yF);
		
		int index = y * srcLinesizeY + x; //index in source image
		int patternIndexX = j % patternXSize;
		int patternIndexY = i % patternYSize;
		float* rowPatternX = patternX[patternIndexX];
		float* rowPatternY = patternY[patternIndexY];
		outputY[i * dstWidth + j] = calculateAreaInterpolation(inputY, index, xRatio, yRatio, srcLinesizeY, 1, rowPatternX, rowPatternY);
		//we should take chroma for every 2 luma, also height of data[1] is twice less than data[0]
		//there are no difference between x_ratio for Y and UV also as for y_ratio because (src_height / 2) / (dst_height / 2) = src_height / dst_height
		if (i < dstHeight / 2 && j < dstWidth / 2) {
			index = y * srcLinesizeUV + x * 2; //index in source image
			int indexU, indexV;
			indexU = index;
			indexV = index + 1;
			outputUV[i * dstWidth + 2 * j] = calculateAreaInterpolation(inputUV, indexU, xRatio, yRatio, srcLinesizeUV, 2, rowPatternX, rowPatternY);
			outputUV[i * dstWidth + 2 * j + 1] = calculateAreaInterpolation(inputUV, indexV, xRatio, yRatio, srcLinesizeUV, 2, rowPatternX, rowPatternY);
		}
	}
}

__global__ void resizeNV12UpscaleAreaKernel(unsigned char* inputY, unsigned char* inputUV, unsigned char* outputY, unsigned char* outputUV,
	int srcWidth, int srcHeight, int srcLinesizeY, int srcLinesizeUV, int dstWidth, int dstHeight, float xRatio, float yRatio) {
	unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; //coordinate of pixel (y) in destination image
	unsigned int j = blockIdx.x * blockDim.x + threadIdx.x; //coordinate of pixel (x) in destination image

	if (i < dstHeight && j < dstWidth) {
		int x = floorf(xRatio * j); //it's coordinate of pixel in source image
		float xFloat = (j + 1) - (x + 1) / xRatio;
		if (xFloat <= 0)
			xFloat = 0;
		else
			xFloat = xFloat - floorf(xFloat);

		int y = floorf(yRatio * i); //it's coordinate of pixel in source image
		float yFloat = (i + 1) - (y + 1) / yRatio;
		if (yFloat <= 0)
			yFloat = 0;
		else
			yFloat = yFloat - floorf(yFloat);

		outputY[i * dstWidth + j] = calculateBillinearInterpolation(inputY, x, y, 1, 1, srcLinesizeY, srcWidth, srcHeight, xFloat, yFloat);
		if (i < dstHeight / 2 && j < dstWidth / 2) {
			outputUV[i * dstWidth + 2 * j] = calculateBillinearInterpolation(inputUV, 2 * x, y, 2, 1, srcLinesizeUV, srcWidth, srcHeight / 2, xFloat, yFloat);
			outputUV[i * dstWidth + 2 * j + 1] = calculateBillinearInterpolation(inputUV, 2 * x + 1, y, 2, 1, srcLinesizeUV, srcWidth, srcHeight / 2, xFloat, yFloat);
		}
	}
}

__global__ void resizeNV12NearestKernel(unsigned char* inputY, unsigned char* inputUV, unsigned char* outputY, unsigned char* outputUV,
	int srcWidth, int srcHeight, int srcLinesizeY, int srcLinesizeUV, int dstWidth, int dstHeight, float xRatio, float yRatio) {

	unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; //coordinate of pixel (y) in destination image
	unsigned int j = blockIdx.x * blockDim.x + threadIdx.x; //coordinate of pixel (x) in destination image

	if (i < dstHeight && j < dstWidth) {
		int y = (int)(yRatio * i); //it's coordinate of pixel in source image
		int x = (int)(xRatio * j); //it's coordinate of pixel in source image
		/*
		Bit to bit with not biased approach
		float yF = (float)((i + 0.5f) * yRatio - 0.5f); //it's coordinate of pixel in source image
		float xF = (float)((j + 0.5f) * xRatio - 0.5f); //it's coordinate of pixel in source image
		int x = floor(xF);
		int y = floor(yF);
		*/
		int index = y * srcLinesizeY + x; //index in source image
		outputY[i * dstWidth + j] = inputY[index];
		//we should take chroma for every 2 luma, also height of data[1] is twice less than data[0]
		//there are no difference between x_ratio for Y and UV also as for y_ratio because (src_height / 2) / (dst_height / 2) = src_height / dst_height
		if (i < dstHeight / 2 && j < dstWidth / 2) {
			outputUV[i * dstWidth + 2 * j] = inputUV[y * srcLinesizeUV + 2 * x];
			outputUV[i * dstWidth + 2 * j + 1] = inputUV[y * srcLinesizeUV + 2 * x + 1];
		}
	}
}

__global__ void resizeNV12BilinearKernel(unsigned char* inputY, unsigned char* inputUV, unsigned char* outputY, unsigned char* outputUV,
	int srcWidth, int srcHeight, int srcLinesizeY, int srcLinesizeUV, int dstWidth, int dstHeight, float xRatio, float yRatio) {

	unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; //coordinate of pixel (y) in destination image
	unsigned int j = blockIdx.x * blockDim.x + threadIdx.x; //coordinate of pixel (x) in destination image

	if (i < dstHeight && j < dstWidth) {
		float yF = (float)((i + 0.5f) * yRatio - 0.5f); //it's coordinate of pixel in source image
		float xF = (float)((j + 0.5f) * xRatio - 0.5f); //it's coordinate of pixel in source image
		int x = floorf(xF);
		int y = floorf(yF);
		float weightX = xF - x;
		float weightY = yF - y;
		
		//need to avoid empty lines at the top and left corners
		if (x < 0) {
			x = 0;
			weightX = 0;
		}

		if (y < 0) {
			y = 0;
			weightY = 0;
		}

		if (x > srcWidth - 1) {
			x = srcWidth - 1;
			weightX = 0;
		}

		if (y > srcHeight - 1) {
			y = srcHeight - 1;
			weightY = 0;
		}

		outputY[i * dstWidth + j] = calculateBillinearInterpolation(inputY, x, y, 1, 1, srcLinesizeY, srcWidth, srcHeight, weightX, weightY);
		//we should take chroma for every 2 luma, also height of data[1] is twice less than data[0]
		//there are no difference between x_ratio for Y and UV also as for y_ratio because (src_height / 2) / (dst_height / 2) = src_height / dst_height
		if (i < dstHeight / 2 && j < dstWidth / 2) {
			outputUV[i * dstWidth + 2 * j] = calculateBillinearInterpolation(inputUV, 2 * x, y, 2, 1, srcLinesizeUV, srcWidth, srcHeight / 2, weightX, weightY);
			outputUV[i * dstWidth + 2 * j + 1] = calculateBillinearInterpolation(inputUV, 2 * x + 1, y, 2, 1, srcLinesizeUV, srcWidth, srcHeight / 2, weightX, weightY);
		}
	}
}

__global__ void resizeNV12BicubicKernel(unsigned char* inputY, unsigned char* inputUV, unsigned char* outputY, unsigned char* outputUV,
	int srcWidth, int srcHeight, int srcLinesizeY, int srcLinesizeUV, int dstWidth, int dstHeight, float xRatio, float yRatio) {

	unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; //coordinate of pixel (y) in destination image
	unsigned int j = blockIdx.x * blockDim.x + threadIdx.x; //coordinate of pixel (x) in destination image

	if (i < dstHeight && j < dstWidth) {
		double yF = (double)((i + 0.5f) * yRatio - 0.5f); //it's coordinate of pixel in source image
		double xF = (double)((j + 0.5f) * xRatio - 0.5f); //it's coordinate of pixel in source image
		int x = floor(xF);
		int y = floor(yF);
		double weightX = xF - x;
		double weightY = yF - y;

		//need to avoid empty lines at the top and left corners
		if (x < 0) {
			x = 0;
			weightX = 0;
		}

		if (y < 0) {
			y = 0;
			weightY = 0;
		}

		if (x > srcWidth - 1) {
			x = srcWidth - 1;
			weightX = 0;
		}

		if (y > srcHeight - 1) {
			y = srcHeight - 1;
			weightY = 0;
		}

		outputY[i * dstWidth + j] = calculateBicubicSplineInterpolation(inputY, x, y, 1, 1, srcLinesizeY, srcWidth, srcHeight, weightX, weightY);
		//we should take chroma for every 2 luma, also height of data[1] is twice less than data[0]
		//there are no difference between x_ratio for Y and UV also as for y_ratio because (src_height / 2) / (dst_height / 2) = src_height / dst_height
		if (i < dstHeight / 2 && j < dstWidth / 2) {
			outputUV[i * dstWidth + 2 * j] = calculateBicubicSplineInterpolation(inputUV, 2 * x, y, 2, 1, srcLinesizeUV, srcWidth, srcHeight / 2, weightX, weightY);
			outputUV[i * dstWidth + 2 * j + 1] = calculateBicubicSplineInterpolation(inputUV, 2 * x + 1, y, 2, 1, srcLinesizeUV, srcWidth, srcHeight / 2, weightX, weightY);
		}
	}
}

void generateResizePattern(float scale, std::vector<std::vector<float> >& pattern) {
	int currentID = 0;
	float rest = 0;

	while (currentID * scale == 0 || (currentID * scale - (int)(currentID * scale) > std::numeric_limits<float>::epsilon())) {
		float dynScale = scale;
		pattern.push_back(std::vector<float>());
		if (rest) {
			pattern[currentID].push_back(rest);
			dynScale -= rest;
		}
		while (dynScale - 1 > 0) {
			pattern[currentID].push_back(1);
			dynScale--;
		}

		if (dynScale > std::numeric_limits<float>::epsilon()) {
			//push rest to pattern
			pattern[currentID].push_back(dynScale);
			rest = 1 - dynScale;
		}

		while (pattern[currentID].size() < ceil(scale))
			pattern[currentID].push_back(0);

		currentID += 1;
	}
}


float** copy2DArray(std::vector<std::vector<float> > pattern, float ratio) {
	float** patternCUDA;
	hipError_t err = hipMalloc((void **)&patternCUDA, sizeof(float*) * pattern.size());
	float** tempPatternCUDA = (float**)malloc(sizeof(float*) * pattern.size());
	for (int i = 0; i < pattern.size(); i++) {
		err = hipMalloc((void**)&tempPatternCUDA[i], sizeof(float) * (round(ratio) + 1));
		err = hipMemcpy(tempPatternCUDA[i], pattern[i].data(), sizeof(float) * (round(ratio) + 1), hipMemcpyHostToDevice);
	}

	err = hipMemcpy(patternCUDA, tempPatternCUDA, sizeof(float*) * pattern.size(), hipMemcpyHostToDevice);
	return patternCUDA;
}

hipError_t free2DArray(float** pattern, int size, float ratio) {
	hipError_t err;
	err = hipFree(pattern);
	return err;
}

int resizeKernel(AVFrame* src, AVFrame* dst, bool crop, ResizeOptions resize, int maxThreadsPerBlock, hipStream_t * stream) {
	unsigned char* outputY = nullptr;
	unsigned char* outputUV = nullptr;
	hipError_t err = hipMalloc(&outputY, resize.width * resize.height * sizeof(unsigned char)); //in resize we don't change color format
	err = hipMalloc(&outputUV, resize.width * (resize.height / 2) * sizeof(unsigned char));
	//need to execute for width and height
	dim3 threadsPerBlock(64, maxThreadsPerBlock / 64);
	int blockX = std::ceil(resize.width / (float)threadsPerBlock.x);
	int blockY = std::ceil(resize.height / (float)threadsPerBlock.y);
	dim3 numBlocks(blockX, blockY);
	float xRatio = (float)(src->width) / resize.width; //if not -1 we should examine 2x2 square with top-left corner in the last pixel of src, so it's impossible
	float yRatio = (float)(src->height) / resize.height;
	int pitchY = src->linesize[0] ? src->linesize[0] : src->width;
	int pitchUV = src->linesize[1] ? src->linesize[1] : src->width;
	switch (resize.type) {
	case ResizeType::BILINEAR:
		resizeNV12BilinearKernel << <numBlocks, threadsPerBlock, 0, *stream >> > (src->data[0], src->data[1], outputY, outputUV,
			src->width, src->height, pitchY, pitchUV,
			resize.width, resize.height, xRatio, yRatio);
		break;
	case ResizeType::NEAREST:
		resizeNV12NearestKernel << <numBlocks, threadsPerBlock, 0, *stream >> > (src->data[0], src->data[1], outputY, outputUV,
			src->width, src->height, pitchY, pitchUV,
			resize.width, resize.height, xRatio, yRatio);
		break;
	case ResizeType::AREA:
		//The smart "area" algorithm is used only in case of downscaling
		if (xRatio > 1 && yRatio > 1) {
			std::vector<std::vector<float> > patternX;
			std::vector<std::vector<float> > patternY;
			generateResizePattern(xRatio, patternX);
			generateResizePattern(yRatio, patternY);

			float** patternXCUDA = copy2DArray(patternX, xRatio);
			float** patternYCUDA = copy2DArray(patternY, yRatio);

			//Here we should decide which AREA algorithm to use
			resizeNV12DownscaleAreaKernel << <numBlocks, threadsPerBlock, 0, *stream >> > (src->data[0], src->data[1], outputY,
				outputUV, src->width, src->height, pitchY, pitchUV, resize.width, resize.height, xRatio, yRatio,
				patternXCUDA, patternX.size(), patternYCUDA, patternY.size());

			free2DArray(patternXCUDA, patternX.size(), xRatio);
			free2DArray(patternYCUDA, patternY.size(), yRatio);
		}
		//otherwise bilinear algorithm with some weight adjustments is used
		else {
			resizeNV12UpscaleAreaKernel << <numBlocks, threadsPerBlock, 0, *stream >> > (src->data[0], src->data[1], outputY,
				outputUV, src->width, src->height, pitchY, pitchUV, resize.width, resize.height, xRatio, yRatio);
		}
		break;
	case ResizeType::BICUBIC:
		resizeNV12BicubicKernel << <numBlocks, threadsPerBlock, 0, *stream >> > (src->data[0], src->data[1], outputY, outputUV,
			src->width, src->height, pitchY, pitchUV,
			resize.width, resize.height, xRatio, yRatio);
		break;
	}

	if (crop) {
		err = hipFree(dst->data[0]);
		err = hipFree(dst->data[1]);
	}

	dst->data[0] = outputY;
	dst->data[1] = outputUV;
	return err;
}
